#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <iostream>
#include <algorithm>
#include <cmath>
#include "profiler.h"

struct func
{
	__host__ __device__ double operator()(double x, double y)
	{
		return sin(x) + cos(y)*cos(y);
	}
};

using namespace thrust::placeholders;

int main()
{
    int n = 1024 * 1024 * 128;

    profiler prof;

    std::vector<double> a(n);
    std::vector<double> b(n);
    std::vector<double> c_cpu(n);
    std::vector<double> c_gpu(n);

    std::uniform_real_distribution<double> distribution(0.0, 1.0);
    std::mt19937 engine; // Mersenne twister MT19937
    auto generator = std::bind(distribution, engine);

    std::generate_n(a.begin(), n, generator);
    std::generate_n(b.begin(), n, generator);

    thrust::device_vector<double> X(a.begin(), a.end());
    thrust::device_vector<double> Y(b.begin(), b.end());
    thrust::device_vector<double> Z(n);

    prof.tic("thrust");
    for(auto i = 0; i < 100; ++i)
        thrust::transform( X.begin(), X.end(), Y.begin(), Z.begin(), []__device__(double x, double y){ return sin(x) + cos(y) * cos(y); } );
        //thrust::transform( X.begin(), X.end(), Y.begin(), Z.begin(), func() );
        //thrust::transform( X.begin(), X.end(), Y.begin(), Z.begin(), _1 + _2 * _2);
    prof.toc("thrust");

    thrust::copy(Z.begin(), Z.end(), c_gpu.begin());

    prof.tic("cpu sum");
    for(auto i = 0; i < n; ++i)
        c_cpu[i] = sin(a[i]) + cos(b[i])*cos(b[i]);
    prof.toc("cpu sum");


    for(auto i = 0; i < n; ++i)
        if( fabs( c_cpu[i] - c_gpu[i] ) > 1e-5)
        {
            std::cout << "Wrong calculation" << std::endl;
            return 1;
        }

    prof.report();

    return 0;
}
