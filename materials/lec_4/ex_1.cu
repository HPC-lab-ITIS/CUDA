#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <mpi.h>

__global__ void kernel(int i, double *a)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;

    a[idx] = i;
}

int main(int argc, char *argv[])
{
    int rank; //process rank
    int size; //number of processes

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    auto count = 0;
    auto n = 1024;
    auto n_bytes = n * sizeof(double);
    
    hipGetDeviceCount(&count);

    double *a;

    hipSetDevice(rank);

    hipMallocManaged( (void **)&a, n_bytes );

    kernel<<<n / 256, 256>>>(rank, a);

    hipDeviceSynchronize();

    std::cout << std::accumulate(a, a + n, 0.0) << std::endl;

    hipFree(a);

    MPI_Finalize();

    return 0;
}
