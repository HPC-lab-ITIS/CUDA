#include <iostream>
#include <vector>
#include "profiler.h"
#include <algorithm>
#include <hipblas.h>


int main()
{
    const auto n = 208*32;
    std::vector<double> a(n*n,0.);
    std::vector<double> b(n*n,0.);
    std::vector<double> c(n*n,0.);
    std::vector<double> c_host(n*n,0.);
    profiler prof;
    std::uniform_real_distribution<double> distribution(0.0, 1.0);
    std::mt19937 engine; // Mersenne twister MT19937
    auto generator = std::bind(distribution, engine);
    std::generate_n(a.begin(), n, generator);
    std::generate_n(b.begin(), n, generator);
    auto n_bytes = n * n * sizeof(double);
    double *a_dev = nullptr, *b_dev = nullptr, *c_dev = nullptr;

    hipError_t cuerr = hipMalloc ( (void**)&a_dev, n_bytes );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot allocate GPU memory for a_dev" << hipGetErrorString(cuerr);
        return 1;
    }

    cuerr = hipMalloc ( (void**)&b_dev, n_bytes );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot allocate GPU memory for b_dev" << hipGetErrorString(cuerr);
        return 1;
    }

    cuerr = hipMalloc ( (void**)&c_dev, n_bytes );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot allocate GPU memory for c_dev" << hipGetErrorString(cuerr);
        return 1;
    }

    cuerr = hipMemcpy ( a_dev, a.data(), n_bytes, hipMemcpyHostToDevice );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot copy data from a to a_dev" << hipGetErrorString(cuerr);
        return 1;
    }

    cuerr = hipMemcpy ( b_dev, b.data(), n_bytes, hipMemcpyHostToDevice );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot copy data from a to b_dev" << hipGetErrorString(cuerr);
        return 1;
    }
   
    hipblasHandle_t handle;
    hipblasStatus_t cberr = hipblasCreate(&handle);
    if (cberr != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "Cannot create cublas handle: " << cberr << std::endl;
        return 1;
    }

    prof.tic("cublas");
    // Выполнить умножение матриц cdev := adev * bdev на GPU.
    double alpha = 1.0, beta = 0.0;
    cberr = hipblasDgemm(
            handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n,
            &alpha, a_dev, n, b_dev, n, &beta, c_dev, n);
    prof.toc("cublas");

    if (cberr != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "Error launching hipblasSgemm: " << cberr << std::endl;
        return 1;
    }
    // Ожидать завершения операции.
    cuerr = hipDeviceSynchronize();
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot synchronize kernel: " << hipGetErrorString(cuerr) << std::endl;
        return 1;
    }
    // Удалить дексриптор CUBLAS.
    cberr = hipblasDestroy(handle);
    if (cberr != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "Cannot destroy cublas handle: " << cberr << std::endl;
        return 1;
    }

    prof.report();

    return 0;
}
