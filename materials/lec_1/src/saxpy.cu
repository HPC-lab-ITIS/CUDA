
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <functional>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <chrono>

__global__ void saxpy_kernel(float a, float *x, float *y, float *z)
{
	// Вычисляем глобальный индекс нити
	size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

	// Обработка соответствующих каждой нити данных
        z[idx] = a * x[idx] + y[idx];
}

int saxpy_wrapper(std::vector<float> &x, std::vector<float> &y, std::vector<float> &z, float a)
{
    int n = x.size();
    auto n_bytes = n * sizeof(float);
    float *x_dev = nullptr, *y_dev = nullptr, *z_dev = nullptr;

    //Выделить память на GPU для x_dev
    hipError_t cuerr = hipMalloc( (void**)&x_dev, n_bytes );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot allocate GPU memory for x_dev" << hipGetErrorString(cuerr);
        return 1;
    }

    //Выделить память на GPU для н_dev
    cuerr = hipMalloc( (void**)&y_dev, n_bytes );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot allocate GPU memory for y_dev" << hipGetErrorString(cuerr);
        return 1;
    }

    //Выделить память на GPU для z_dev
    cuerr = hipMalloc( (void**)&z_dev, n_bytes );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot allocate GPU memory for z_dev" << hipGetErrorString(cuerr);
        return 1;
    }

    //Задать конфигурацию запуска блоков нитей и сетки блоков
    int block_size = 1024;
    int grid_size = n / block_size;

    //Скопировать входные данные из памяти CPU в память GPU.
    cuerr = hipMemcpy(x_dev, x.data(), n_bytes, hipMemcpyHostToDevice );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot copy data from x to x_dev" << hipGetErrorString(cuerr);
        return 1;
    }

    //Скопировать входные данные из памяти CPU в память GPU.
    cuerr = hipMemcpy(y_dev, y.data(), n_bytes, hipMemcpyHostToDevice );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot copy data from y to y_dev" << hipGetErrorString(cuerr);
        return 1;
    }

    //Создать события для замерения времени
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    // Вызвать ядро с заданной конфигурацией для обработки данных в цикле
    saxpy_kernel<<<grid_size, block_size>>>(a, x_dev, y_dev, z_dev);

    cuerr = hipGetLastError();
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot launch CUDA kernel " << hipGetErrorString(cuerr);
        return 1;
    }

    // Ожидать завершения работы ядра.
    cuerr = hipDeviceSynchronize();
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot synchronize CUDA kernel " << hipGetErrorString(cuerr);
        return 1;
    }
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    //Вывести время исполнения в мс
    float gpu_time = 0.0f;
    hipEventElapsedTime(&gpu_time, start, stop);
    std::cout << "Elapsed time gpu: " << gpu_time << " ms." << std::endl;

    // Скопировать результаты в память CPU.
    cuerr = hipMemcpy(z.data(), z_dev, n_bytes, hipMemcpyDeviceToHost );
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot copy data from z_dev to z " << hipGetErrorString(cuerr);
        return 1;
    }

    // Освободить выделенную память GPU.
    hipFree(x_dev);
    hipFree(y_dev);
    hipFree(z_dev);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

int main(int argc, char *argv[])
{
    size_t n = 1 << 27;

    std::vector<float> x(n);
    std::vector<float> y(n);
    std::vector<float> z_cpu(n);
    std::vector<float> z_gpu(n);

    std::uniform_real_distribution<> distribution(0.0, 1.0);
    std::mt19937 engine; 
    auto generator = std::bind(distribution, engine);
    std::generate_n(x.begin(), n, generator);
    std::generate_n(y.begin(), n, generator);
    float a = distribution(engine);

    auto begin = std::chrono::steady_clock::now();
    for(auto i = 0; i < n; ++i)
        z_cpu[i] = a * x[i] + y[i];
    auto end = std::chrono::steady_clock::now();
    
    std::cout << "Elapsed time cpu: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << " ms." << std::endl;

    saxpy_wrapper(x, y, z_gpu, a);

    for(auto i = 0; i < n; ++i)
        if( fabs( z_cpu[i] - z_gpu[i] ) > 1e-5)
        {
            std::cout << "Wrong calculation" << std::endl;
            return 1;
        }

    return 0;
}
