#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm> 
#include "profiler.h"
__global__ void sum_kernel(size_t n, double *a, double *b, double *c)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t grid_size = blockDim.x * gridDim.x;
    for(size_t i = idx; i < n; i += grid_size)
        c[i] = sin(a[i]) + cos(b[i])*cos(b[i]);
}
 
int main() 
{
    const size_t n = 1024*1024*128;
    double* a; 
    double* b; 
    double* c; 
    hipMallocManaged(&a, n * sizeof(double));
    hipMallocManaged(&b, n * sizeof(double));
    hipMallocManaged(&c, n * sizeof(double));

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dis(0., 1.);

    for (auto i = 0; i < n; i++)
    {
        a[i] = dis(gen);        
        b[i] = dis(gen);
    }

    profiler prof;
    prof.tic("Sum");
    hipDeviceSynchronize();
    for(auto i = 0; i < 100; ++i)
        sum_kernel<<<208, 256>>>(n, a, b, c);
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    prof.toc("Sum");

    hipFree(a);
    hipFree(b);
    hipFree(c);

    prof.report();
    return 0;
}
