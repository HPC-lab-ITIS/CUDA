#include <iostream>
#include "profiler.h"
#include <vector>

int main()
{
    size_t n = 128*1024;
    size_t n_bytes = n*sizeof(double);
    double *a_dev = nullptr, *buffer = nullptr;
    std::vector<double> a_host(n,0.);
    profiler prof;

    hipHostMalloc( (void **) &buffer, n_bytes);

    for(auto i = 0; i < n; ++i)
    {
        a_host[i] = i;
        buffer[i] = i;
    }

    hipError_t cuerr = hipMalloc( (void**)&a_dev, n_bytes);
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot allocate GPU memory" << hipGetErrorString(cuerr);
        return 1;
    }    

    prof.tic("Pinned memory");
    hipDeviceSynchronize();
    for(auto i = 0; i < 1000; ++i)
    {    
        cuerr = hipMemcpy( a_dev, buffer, n_bytes, hipMemcpyHostToDevice );

        if (cuerr != hipSuccess)
        {
            std::cout << "Cannot copy data to device" << hipGetErrorString(cuerr);
            return 1;
        }

        cuerr = hipMemcpy ( buffer, a_dev, n_bytes, hipMemcpyDeviceToHost );
        if (cuerr != hipSuccess)
        {
            std::cout << "Cannot copy data from device" << hipGetErrorString(cuerr);
            return 1;
        }
    }
    hipDeviceSynchronize();
    prof.toc("Pinned memory");

    prof.tic("Paged memory");
    hipDeviceSynchronize();
    for(auto i = 0; i < 1000; ++i)
    {    
        cuerr = hipMemcpy( a_dev, a_host.data(), n_bytes, hipMemcpyHostToDevice );

        if (cuerr != hipSuccess)
        {
            std::cout << "Cannot copy data to device" << hipGetErrorString(cuerr);
            return 1;
        }

        cuerr = hipMemcpy ( a_host.data(), a_dev, n_bytes, hipMemcpyDeviceToHost );
        if (cuerr != hipSuccess)
        {
            std::cout << "Cannot copy data from device" << hipGetErrorString(cuerr);
            return 1;
        }
    }
    hipDeviceSynchronize();
    prof.toc("Paged memory");

    hipFree(a_dev);
    hipHostFree(buffer);

    prof.report();

    return 0;
}
