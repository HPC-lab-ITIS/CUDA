#include "hip/hip_runtime.h"
#include <iostream>
#include "profiler.h"
#include <vector>

__global__ void kernel(double *a)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	a[i] += i;
}


int main()
{
    size_t n = 128*1024;
    size_t n_bytes = n*sizeof(double);
    double *a_dev = nullptr, *buffer = nullptr, *buff_map = nullptr;
    std::vector<double> a_host(n,0.);
    profiler prof;

    hipSetDeviceFlags(hipDeviceMapHost); 
    hipHostAlloc(&buffer, n_bytes, hipHostMallocMapped);

    for(auto i = 0; i < n; ++i)
    {
        a_host[i] = i;
        buffer[i] = i;
    }

    hipHostGetDevicePointer(&buff_map, buffer, 0);
    
    hipError_t cuerr = hipMalloc( (void**)&a_dev, n_bytes);
    if (cuerr != hipSuccess)
    {
        std::cout << "Cannot allocate GPU memory" << hipGetErrorString(cuerr);
        return 1;
    }    

    prof.tic("Zero copy");
    hipDeviceSynchronize();
    for(auto i = 0; i < 1000; ++i)
    {    
        kernel<<<13, 256>>>(buff_map);
        cuerr = hipGetLastError();
        if (cuerr != hipSuccess)
        {
            std::cout << "Cannot launch CUDA kernel " << hipGetErrorString(cuerr);
            return 1;
        }

        cuerr = hipDeviceSynchronize();
        if (cuerr != hipSuccess)
        {
            std::cout << "Cannot synchronize CUDA kernel " << hipGetErrorString(cuerr);
            return 1;
        }

    }
    hipDeviceSynchronize();
    prof.toc("Zero copy");

    prof.tic("Standart copy");
    hipDeviceSynchronize();
    for(auto i = 0; i < 1000; ++i)
    {    
        cuerr = hipMemcpy( a_dev, a_host.data(), n_bytes, hipMemcpyHostToDevice );

        if (cuerr != hipSuccess)
        {
            std::cout << "Cannot copy data to device" << hipGetErrorString(cuerr);
            return 1;
        }

        kernel<<<13, 256>>>(a_dev);
        cuerr = hipGetLastError();
        if (cuerr != hipSuccess)
        {
            std::cout << "Cannot launch CUDA kernel " << hipGetErrorString(cuerr);
            return 1;
        }

        cuerr = hipMemcpy ( a_host.data(), a_dev, n_bytes, hipMemcpyDeviceToHost );
        if (cuerr != hipSuccess)
        {
            std::cout << "Cannot copy data from device" << hipGetErrorString(cuerr);
            return 1;
        }
    }
    hipDeviceSynchronize();
    prof.toc("Standart copy");

    for(auto i = 0; i < n; ++i)
        if(fabs(a_host[i] - buffer[i])>1e-5)
        {
            std::cout << "fail" << std::endl;
            return 1;
        }

    hipFree(a_dev);
    hipHostFree(buffer);

    prof.report();

    return 0;
}
