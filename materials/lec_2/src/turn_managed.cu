#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <functional>
#include <fstream>
#include <sstream>
#include <algorithm>
#include "profiler.h"
#include "turn.cuh"


int main()
{
    profiler prof;
    const size_t n = 1 << 27;
    float *x = nullptr, *y = nullptr, *tmp = nullptr, *angle = nullptr;

    hipMallocManaged(&x, n * sizeof(float));
    hipMallocManaged(&y, n * sizeof(float));
    hipMallocManaged(&tmp, 4 * sizeof(float));
    hipMallocManaged(&angle, sizeof(float));

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0., 1.);

    for (auto i = 0; i < n; i++)
    {
        x[i] = dis(gen);        
        y[i] = dis(gen);
    }
    (*angle) = 45.;

    tmp[0] = cos((*angle));
    tmp[1] = -sin((*angle));
    tmp[2] = sin((*angle));
    tmp[3] = cos((*angle));

    prof.tic("cpu turn");
#pragma omp parallel for num_threads(8)    
    for(int i = 0; i < n; ++i)
    {
        float tmp_x = x[i] * cos(*angle) - y[i] * sin(*angle);
        float tmp_y = x[i] * sin(*angle) + y[i] * cos(*angle);

        x[i] = tmp_x;
        y[i] = tmp_y;
    }
    prof.toc("cpu turn");

    int threads = 1024;
    int blocks = n / threads;
    
    prof.tic("gpu turn global memory");
    for(int i = 0; i < 10; ++i)
    {
        turn_kernel<<<blocks, threads>>>(x, y, tmp);
        cudaCheckError( hipGetLastError() );
        cudaCheckError( hipDeviceSynchronize() );
    }
    prof.toc("gpu turn global memory");
    
    prof.tic("gpu turn constant memory");
    for(int i = 0; i < 10; ++i)
    {
        turn_kernel<<<blocks, threads>>>(x, y);
        cudaCheckError( hipGetLastError() );
        cudaCheckError( hipDeviceSynchronize() );
    }
    prof.toc("gpu turn constant memory");

    hipFree(x);
    hipFree(y);
    hipFree(tmp);
    hipFree(angle);

    prof.report();

    return 0;
}
