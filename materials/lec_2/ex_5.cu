struct dataElem {
  int prop1;
  int prop2;
  char *name;
}

void launch(dataElem *elem)
{
  dataElem *d_elem;
  char *d_name;

  int namelen = strlen(elem->name) + 1;

  // Выделяем память под структуру и под поле name
  hipMalloc(&d_elem, sizeof(dataElem));
  hipMalloc(&d_name, namelen);

  // Отдельно копируем структуру, значение поля name и значение указателя на поле name 
  hipMemcpy(d_elem, elem, sizeof(dataElem), hipMemcpyHostToDevice);
  hipMemcpy(d_name, elem->name, namelen, hipMemcpyHostToDevice);
  hipMemcpy(&(d_elem->name), &d_name, sizeof(char*), hipMemcpyHostToDevice);

  // Уффф
  Kernel<<< ... >>>(d_elem);
}

//Вместо 1000 слов
void launch_UV(dataElem *elem)
{
  kernel<<< ... >>>(elem);
}

/*-----------------------*/
class Managed 
{
public:
  void *operator new(size_t len)
  {
    void *ptr;
    hipMallocManaged(&ptr, len);
    hipDeviceSynchronize();
    return ptr;
  }

  void operator delete(void *ptr) 
  {
    hipDeviceSynchronize();
    hipFree(ptr);
  }
};

// Наследование от Managed позволяет ссылку по указателю
class String : public Managed 
{
  int length;
  char *data;

public:
  // Unified memory позволяет конструктор копирования
  String (const String &s) 
  {
    length = s.length;
    hipMallocManaged(&data, length);
    memcpy(data, s.data, length);
  }

  // ...
};